#include "hip/hip_runtime.h"
﻿#include <iostream>
#include <cstdlib>
#include <ctime>
#include <hip/hip_runtime.h>
#include ""
#include <omp.h>


__global__ void matrixMultiplyKernel(int* A, int* B, int* result, int n, int m, int k) {
    int row = blockIdx.y * blockDim.y + threadIdx.y;
    int col = blockIdx.x * blockDim.x + threadIdx.x;

    if (row < n && col < n) {
        int value = 0;
        for (int i = 0; i < m; i++) {
            value += A[row * m + i] * B[i * k + col];
        }
        result[row * n + col] = value;
    }
}


void initializeMatrix(int* matrix, int rows, int cols) {
    for (int i = 0; i < rows; i++) {
        for (int j = 0; j < cols; j++) {
            matrix[i * cols + j] = rand() % 10 + 1;
        }
    }
}

int main() {
    srand(static_cast<unsigned>(time(0)));

    hipSetDevice(0);
    for(int n = 32; n<=16384; n = n*2)
    {
                
                std::cout << "n: " << n << std::endl;

                int m = n;
                int* A = new int[n * m];
                int* B = new int[m * n];
                int* resultCUDA = new int[n * n];


                initializeMatrix(A, n, m);
                initializeMatrix(B, m, n);

                int* d_A, * d_B, * d_result;
                hipMalloc(&d_A, n * m * sizeof(int));
                hipMalloc(&d_B, m * n * sizeof(int));
                hipMalloc(&d_result, n * n * sizeof(int));

                hipMemcpy(d_A, A, n * m * sizeof(int), hipMemcpyHostToDevice);
                hipMemcpy(d_B, B, m * n * sizeof(int), hipMemcpyHostToDevice);

                dim3 blockSize(16, 16);
                dim3 gridSize((n + blockSize.x - 1) / blockSize.x, (n + blockSize.y - 1) / blockSize.y);

                clock_t cudaStart = clock();
                matrixMultiplyKernel << <gridSize, blockSize >> > (d_A, d_B, d_result, n, m, n);
                hipDeviceSynchronize();
                clock_t cudaEnd = clock();

                std::cout << "CUDA algo run time: " << 1000.0 * (cudaEnd - cudaStart) / CLOCKS_PER_SEC << " ms" << std::endl;

                hipMemcpy(resultCUDA, d_result, n * n * sizeof(int), hipMemcpyDeviceToHost);

                //double speedup = (sequentialEnd - sequentialStart) / (cudaEnd - cudaStart);
                //std::cout << "Speedup: " << speedup << std::endl;

                delete[] A;
                delete[] B;
                delete[] resultCUDA;

                hipFree(d_A);
                hipFree(d_B);
                hipFree(d_result);

    }
        return 0;
}
